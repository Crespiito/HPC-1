#include "hip/hip_runtime.h"
#include "opencv2/opencv.hpp"
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace cv;


__global__void sobelFilter(unsigned char * d_imagegray, unsigned char *d_imagefiltered, int width, int height){

	int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    int limitRow = height - 1, limitCol = width - 1, *sobelMaskRow, *sobelMaskCol;
    
    sobelMaskRow= (int*)malloc(9*sizeof(int));
    sobelMaskCol = (int*)malloc(9*sizeof(int));

    sobelMaskRow[0] = 1; sobelMaskRow[1] = 0; sobelMaskRow[2] = -1;
    sobelMaskRow[3] = 2; sobelMaskRow[4] = 0; sobelMaskRow[5] = -2;
    sobelMaskRow[6] = 1; sobelMaskRow[7] = 0; sobelMaskRow[8] = -1;

    sobelMaskCol[0] = -1; sobelMaskCol[0] = -2; sobelMaskCol[0] = -1;
    sobelMaskCol[0] = 0; sobelMaskCol[0] = 1; sobelMaskCol[0] = 0;
    sobelMaskCol[0] = 1; sobelMaskCol[0] = 2; sobelMaskCol[0] = 1;

    for (int i = 0; i < 3; ++i){
        for (int i = 0; i < count; ++i){
            if (limitCol >= 0 && limitRow >= 0 && limitRow < height && limitCol < width){

            }
        }
    }
}

__global__ void imgGray(unsigned char * d_image, unsigned char* d_imagegray, int width, int height){
    
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if ((width > col) && (height > row)){
        d_imagegray[row*width+col]=d_image[(row*width+col)*3+2]*0.3+d_image[(row*width+col)*3+1]*0.6+d_image[(row*width+col)*3]*0.2;
    }
}

int main(int argc, char const *argv[])
{
    uchar *h_image, *d_image, *h_imagegray, *d_imagegray, *h_imagefiltered, *d_imagefiltered;

    Mat image = imread(argv[1],1);
    Size s = image.size();
    int sizeRGB = s.width*s.height*image.channels()*sizeof(unsigned char);
    int sizeGray = s.width*s.height*sizeof(unsigned char);
    int blockSize = 32;
    printf("%d , %d \n",sizeRGB, sizeGray);
    if (image.empty()){
        printf("Not found the image \n");
    }

    h_imagegray = (unsigned char*)malloc(sizeGray);
    hipMalloc((void**)&d_image,sizeRGB);
    hipMalloc((void**)&d_imagegray,sizeGray);

    h_image = image.data;

    hipMemcpy(d_image,h_image,sizeRGB,hipMemcpyHostToDevice);
    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimTrheads(ceil(s.width/float(blockSize)),ceil(s.height/float(blockSize)),1);
    imgGray<<<dimTrheads,dimBlock>>>(d_image,d_imagegray,s.width,s.height);
    hipDeviceSynchronize();
    hipMemcpy(h_imagegray,d_imagegray,sizeGray,hipMemcpyDeviceToHost);

    Mat imageGray;
    imageGray.create(s.height,s.width,CV_8UC1);
    imageGray.data = h_imagegray;
    imwrite("./ImageG.jpg",imageGray);

    return 0;
}

