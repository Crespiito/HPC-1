#include "hip/hip_runtime.h"
#include "opencv2/opencv.hpp"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

using namespace cv;

__device__ unsigned char clamp(int value){
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return (unsigned char)value;
}


__global__ void sobelFilter(unsigned char * d_imagegray, unsigned char *d_imagefiltered, int width, int height, char* MaskRow, char * MaskCol){

    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    int limitRow = height - 1, limitCol = width - 1;
    float tmpR,tmpC;
    int aux_row = row - 1, aux_col = col - 1; 
    
    for (int i = 0; i < 3; ++i){
        for (int j = 0; j < 3; ++j){
            if (limitCol >= 0 && limitRow >= 0 && limitRow < height && limitCol < width){
                tmpR += d_imagegray[aux_row*width + aux_col]*MaskRow[(i*3)+j];
                aux_col += 1;
            }
            aux_row += 1;
            aux_col = col - 1 ;
        }
    }

    aux_row = row - 1, aux_col = col - 1; 
    
    for (int i = 0; i < 3; ++i){
        for (int j = 0; j < 3; ++j){
            if (limitCol >= 0 && limitRow >= 0 && limitRow < height && limitCol < width){
                tmpC += d_imagegray[aux_row*width + aux_col]*MaskCol[(i*3)+j];
                aux_col += 1;
            }
            aux_row += 1;
            aux_col = col - 1 ;
        }
    }
        
    d_imagefiltered[(row * width) + col] = clamp(sqrt(pow(tmpC,2) + pow(tmpR , 2)));
}

__global__ void imgGray(unsigned char * d_image, unsigned char* d_imagegray, int width, int height){
    
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if ((width > col) && (height > row)){
        d_imagegray[row*width+col]=d_image[(row*width+col)*3+2]*0.3+d_image[(row*width+col)*3+1]*0.6+d_image[(row*width+col)*3]*0.2;
    }
}

int main(int argc, char const *argv[])
{
    uchar *h_image, *d_image, *h_imagegray, *d_imagegray, *h_imagefiltered, *d_imagefiltered;
    clock_t start, end; // Medir tiempos

    Mat image = imread(argv[1],1);
    Size s = image.size();
    int sizeRGB = s.width*s.height*image.channels()*sizeof(unsigned char);
    int sizeGray = s.width*s.height*sizeof(unsigned char);
    int blockSize = 32;

    printf("%d , %d \n",sizeRGB, sizeGray);
    if (image.empty()){
        printf("Not found the image \n");
    }

    h_imagegray = (unsigned char*)malloc(sizeGray);
    h_imagefiltered = (unsigned char*)malloc(sizeGray);
    hipMalloc((void**)&d_image,sizeRGB);
    hipMalloc((void**)&d_imagegray,sizeGray);

    h_image = image.data;

    hipMemcpy(d_image,h_image,sizeRGB,hipMemcpyHostToDevice);
    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimTrheads(ceil(s.width/float(blockSize)),ceil(s.height/float(blockSize)),1);

    char h_sobelMaskRow[] = { 1 ,0, -1, 2, 0, -2, 1, 0, -1 };
    char h_sobelMaskCol[] = { -1 , -2, -1, 0, 1, 0, 1, 1, 1};

    char *d_sobelMaskRow;
    char *d_sobelMaskCol;

    hipMalloc((char**)&d_sobelMaskRow,sizeof(char)*9);
    hipMalloc((char**)&d_sobelMaskCol,sizeof(char)*9);
    hipMalloc((void**)&d_imagefiltered,sizeGray);

    hipMemcpy(d_sobelMaskRow,h_sobelMaskRow,sizeof(char)*9,hipMemcpyHostToDevice);
    hipMemcpy(d_sobelMaskCol,h_sobelMaskCol,sizeof(char)*9,hipMemcpyHostToDevice);

    start = clock(); //Inicia reloj
    imgGray<<<dimTrheads,dimBlock>>>(d_image,d_imagegray,s.width,s.height);
    hipDeviceSynchronize();
    hipMemcpy(h_imagegray,d_imagegray,sizeGray,hipMemcpyDeviceToHost);

    sobelFilter<<<dimTrheads,dimBlock>>>(d_imagegray,d_imagefiltered,s.width,s.height,d_sobelMaskRow,d_sobelMaskCol);
    hipDeviceSynchronize();
    hipMemcpy(h_imagefiltered,d_imagefiltered,sizeGray,hipMemcpyDeviceToHost);
    end = clock(); //Finaliza reloj

    Mat imageGray;
    imageGray.create(s.height,s.width,CV_8UC1);
    imageGray.data = h_imagegray;
    imwrite("./ImageG.jpg",imageGray);

    Mat imageSobel;
    imageSobel.create(s.height,s.width,CV_8UC1);
    imageSobel.data = h_imagefiltered;
    imwrite("./ImageS.jpg",imageSobel);

    cout <<"Tiempo:"<<((double)(end-start))/CLOCKS_PER_SEC<<endl;

    return 0;
}

