#include "hip/hip_runtime.h"
#include "opencv2/opencv.hpp"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

using namespace cv;

__constant__ char MaskRow[9];
__constant__ char MaskCol[9];


__device__ unsigned char clamp(int value){
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return (unsigned char)value;
}


__global__ void sobelFilter(unsigned char * d_imagegray, unsigned char *d_imagefiltered, int width, int height){


    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    __shared__ unsigned char window[34][34];

if ((width > col) && (height > row)){
//llenar Linea Superior con 0
    if(row == 0){
            if(col == 0 ){
                window[0][0]=0;
                window[1][0]=0;
            }else if (col > 0 && threadIdx.x == 0 ){
                window[0][0]=0;
                window[1][0]=d_imagegray[row*width+((blockIdx.x-1)*blockDim.x+31)];
            }

            if(col == (width-1)){
                window[0][33]=0;
                window[1][33]=0;
            }else if(threadIdx.x == 31){
                window[0][33]=0;
                window[1][33]=d_imagegray[row*width+((blockIdx.x+1)*blockDim.x)];
            }
            window[threadIdx.y][threadIdx.x+1]=0;
            window[threadIdx.y+1][threadIdx.x+1]=d_imagegray[row*width+col];
    }

//Llenar linea inferior con  0
    if(row == (height-1)){
            if(col == 0 ){
                window[32][0]=0;
                window[33][0]=0;
            }else if (col > 0 && threadIdx.x == 0 ){
                window[32][0]=d_imagegray[row*width+((blockIdx.x-1)*blockDim.x+31)];
                window[33][0]=0;
            }

            if(col == (width-1)){
                window[32][threadIdx.x+2]=0;
                window[33][threadIdx.x+2]=0;
            }else if(threadIdx.x == 31){
                window[33][33]=0;
                window[32][33]=d_imagegray[row*width+((blockIdx.x+1)*blockDim.x)];
            }
            window[threadIdx.y+2][threadIdx.x+1]=0;
            window[threadIdx.y+1][threadIdx.x+1]=d_imagegray[row*width+col];
    }


//Llenar lineas interio	|res
    else if(row > 0 && row < height){
            if(col == 0 ){
                window[threadIdx.y+1][0]=0;
            }else if (col > 0 && threadIdx.x == 0 ){
                window[threadIdx.y+1][0]=d_imagegray[row*width+((blockIdx.x-1)*blockDim.x+31)];
            }
    if(col > 0 && col < width){
            if(col == (width-1)){
                window[threadIdx.y+1][threadIdx.x+2]=0;
            }else if(threadIdx.x == 31){
                window[threadIdx.y+1][threadIdx.x+2]=d_imagegray[row*width+((blockIdx.x+1)*blockDim.x)];
            }

            if (threadIdx.y == 0){
                window[threadIdx.y][threadIdx.x+1]=d_imagegray[((blockIdx.y-1)*blockDim.y+31)*width+col];

            }

	    if (threadIdx.y == 0 && threadIdx.x == 0){
                window[threadIdx.y][threadIdx.x]=d_imagegray[((blockIdx.y-1)*blockDim.y+31)*width+((blockIdx.x-1)*blockDim.x+31)];
            }

            if (threadIdx.y == 31){
                window[threadIdx.y][threadIdx.x+1]=d_imagegray[((blockIdx.y+1)*blockDim.y)*width+col];
            }

	    if (threadIdx.y == 0 && threadIdx.x == 31){
                window[threadIdx.y][threadIdx.x+2]=d_imagegray[((blockIdx.y-1)*blockDim.y+31)*width+((blockIdx.x+1)*blockDim.x)];
            }

	    if (threadIdx.y == 31 && threadIdx.x == 0){
                window[threadIdx.y+2][threadIdx.x]=d_imagegray[((blockIdx.y+1)*blockDim.y)*width+((blockIdx.x-1)*blockDim.x+31)];
            }

	    if (threadIdx.y == 31 && threadIdx.x == 31){
                window[threadIdx.y+2][threadIdx.x]=d_imagegray[((blockIdx.y+1)*blockDim.y)*width+((blockIdx.x+1)*blockDim.x)];
            }
	}
            window[threadIdx.y+1][threadIdx.x+1]=d_imagegray[row*width+col];

    }
}

__syncthreads();

    float tmpR,tmpC;
    int trow = threadIdx.y+1;
    int tcol = threadIdx.x+1;

    int aux_row = trow - 1, aux_col = tcol - 1;

    for (int i = 0; i < 3; ++i){
        for (int j = 0; j < 3; ++j){
            tmpR += (window[aux_row][aux_col])*MaskRow[(i*3)+j];
            aux_col += 1;
        }

        aux_row += 1;
        aux_col = tcol - 1 ;

    }



    aux_row = trow - 1, aux_col = tcol - 1;

    for (int i = 0; i < 3; ++i){
        for (int j = 0; j < 3; ++j){
                tmpC += window[aux_row][aux_col]*MaskCol[(i*3)+j];
                aux_col += 1;
        }
	if(col== 0 &&  row == 0){
		printf(" %d %d " , aux_row , aux_col);
	}
        aux_row += 1;
        aux_col = tcol - 1 ;
    }

    d_imagefiltered[(row * width) + col] = clamp(sqrt(pow(tmpC,2) + pow(tmpR , 2)));
}

__global__ void imgGray(unsigned char * d_image, unsigned char* d_imagegray, int width, int height){

    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if ((width > col) && (height > row)){
        d_imagegray[row*width+col]=d_image[(row*width+col)*3+2]*0.3+d_image[(row*width+col)*3+1]*0.6+d_image[(row*width+col)*3]*0.2;
    }
}

int main(int argc, char const *argv[])
{
    uchar *h_image, *d_image, *h_imagegray, *d_imagegray, *h_imagefiltered, *d_imagefiltered;

    Mat image = imread(argv[1],1);
    Size s = image.size();
    int sizeRGB = s.width*s.height*image.channels()*sizeof(unsigned char);
    int sizeGray = s.width*s.height*sizeof(unsigned char);
    int blockSize = 32;

    printf("%d , %d \n",sizeRGB, sizeGray);
    if (image.empty()){
        printf("Not found the image \n");
    }

    h_imagegray = (unsigned char*)malloc(sizeGray);
    hipMalloc((void**)&d_image,sizeRGB);
    hipMalloc((void**)&d_imagegray,sizeGray);

    h_image = image.data;

    hipMemcpy(d_image,h_image,sizeRGB,hipMemcpyHostToDevice);
    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimTrheads(ceil(s.width/float(blockSize)),ceil(s.height/float(blockSize)),1);
    imgGray<<<dimTrheads,dimBlock>>>(d_image,d_imagegray,s.width,s.height);
    hipDeviceSynchronize();
    hipMemcpy(h_imagegray,d_imagegray,sizeGray,hipMemcpyDeviceToHost);

    char h_sobelMaskRow[] = { 1 ,0, -1, 2, 0, -2, 1, 0, -1 };
    char h_sobelMaskCol[] = { -1 , -2, -1, 0, 1, 0, 1, 1, 1};
    for(int i =0 ; i<sizeGray/2 ; i++){
      printf("pos %d %d\n",i, h_imagegray[i]);
    }

    hipMalloc((void**)&d_imagefiltered,sizeGray);

    hipMemcpyToSymbol(HIP_SYMBOL(MaskRow),h_sobelMaskRow,sizeof(char)*9);
    hipMemcpyToSymbol(HIP_SYMBOL(MaskCol),h_sobelMaskCol,sizeof(char)*9);

    sobelFilter<<<dimTrheads,dimBlock>>>(d_imagegray,d_imagefiltered,s.width,s.height);
    hipDeviceSynchronize();
    h_imagefiltered = (unsigned char*)malloc(sizeGray);
    hipMemcpy(h_imagefiltered,d_imagefiltered,sizeGray,hipMemcpyDeviceToHost);

    Mat imageGray;
    imageGray.create(s.height,s.width,CV_8UC1);
    imageGray.data = h_imagegray;
    imwrite("./ImageG.jpg",imageGray);

    Mat imageSobel;
    imageSobel.create(s.height,s.width,CV_8UC1);
    imageSobel.data = h_imagefiltered;
    imwrite("./ImageS.jpg",imageSobel);

 //liberar memoria 
    return 0;
}
