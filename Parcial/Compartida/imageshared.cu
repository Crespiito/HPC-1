#include "hip/hip_runtime.h"
#include "opencv2/opencv.hpp"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

using namespace cv;

__constant__ char MaskRow[9];
__constant__ char MaskCol[9];


__device__ unsigned char clamp(int value){
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return (unsigned char)value;
}


__global__ void sobelFilter(unsigned char * d_imagegray, unsigned char *d_imagefiltered, int width, int height){


    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    __shared__ unsigned char window[34][34];

if ((width > col) && (height > row)){
//llenar Linea Superior con 0
    if(row == 0){
            if(col == 0 ){
                window[0][0]=0;
                window[1][0]=0;
            }else if (col > 0 && threadIdx.x == 0 ){
                window[0][0]=0;
                window[1][0]=d_imagegray[row*width+((blockIdx.x-1)*blockDim.x+31)];
            }

            if(col == (width-1)){
                window[0][33]=0;
                window[1][33]=0;
            }else if(threadIdx.x == 31){
                window[0][33]=0;
                window[1][33]=d_imagegray[row*width+((blockIdx.x+1)*blockDim.x)];
            }
            window[threadIdx.y][threadIdx.x+1]=255;
            window[threadIdx.y+1][threadIdx.x+1]=d_imagegray[row*width+col];
    }

//Llenar linea inferior con  0
    if(row == (height-1)){
            if(col == 0 ){
                window[32][0]=0;
                window[33][0]=0;
            }else if (col > 0 && threadIdx.x == 0 ){
                window[32][0]=d_imagegray[row*width+((blockIdx.x-1)*blockDim.x+31)];
                window[33][0]=0;
            }

            if(col == (width-1)){
                window[32][threadIdx.x+2]=0;
                window[33][threadIdx.x+2]=0;
            }else if(threadIdx.x == 31){
                window[33][33]=0;
                window[32][33]=d_imagegray[row*width+((blockIdx.x+1)*blockDim.x)];
            }
            window[threadIdx.y+2][threadIdx.x+1]=0;
            window[threadIdx.y+1][threadIdx.x+1]=d_imagegray[row*width+col];
    }


//Llenar lineas interio	|res
    else if(row > 0 && row < height){
            if(col == 0 ){
                window[threadIdx.y+1][0]=0;
            }

    if (col > 0 && threadIdx.x == 0 ){
        window[threadIdx.y+1][0]=d_imagegray[row*width+((blockIdx.x-1)*blockDim.x+31)];
    }

    if(col > 0 && col < width){
            if(col == (width-1)){
                window[threadIdx.y+1][threadIdx.x+2]=0;
            }

	    if(threadIdx.x == 31){
                window[threadIdx.y+1][threadIdx.x+2]=d_imagegray[row*width+((blockIdx.x+1)*blockDim.x)];
            }

            if (threadIdx.y == 0){
                window[threadIdx.y][threadIdx.x+1]=d_imagegray[((blockIdx.y-1)*blockDim.y+31)*width+col];

            }

	    if (threadIdx.y == 0 && threadIdx.x == 0){
                window[threadIdx.y][threadIdx.x]=d_imagegray[((blockIdx.y-1)*blockDim.y+31)*width+((blockIdx.x-1)*blockDim.x+31)];
            }

            if (threadIdx.y == 31){
                window[threadIdx.y+2][threadIdx.x+1]=d_imagegray[((blockIdx.y+1)*blockDim.y)*width+col];
            }

	    if (threadIdx.y == 0 && threadIdx.x == 31){
                window[threadIdx.y][threadIdx.x+2]=d_imagegray[((blockIdx.y-1)*blockDim.y+31)*width+((blockIdx.x+1)*blockDim.x)];
            }

	    if (threadIdx.y == 31 && threadIdx.x == 0){
                window[threadIdx.y+2][threadIdx.x]=d_imagegray[((blockIdx.y+1)*blockDim.y)*width+((blockIdx.x-1)*blockDim.x+31)];
            }

	    if (threadIdx.y == 31 && threadIdx.x == 31){
                window[threadIdx.y+2][threadIdx.x+2]=d_imagegray[((blockIdx.y+1)*blockDim.y)*width+((blockIdx.x+1)*blockDim.x)];
            }
	}
            window[threadIdx.y+1][threadIdx.x+1]=d_imagegray[row*width+col];

    }
}

__syncthreads();

    float tmpR,tmpC;
    int trow = threadIdx.y+1;
    int tcol = threadIdx.x+1;

    int aux_row = trow - 1, aux_col = tcol - 1;

    for (int i = 0; i < 3; ++i){
        for (int j = 0; j < 3; ++j){
            tmpR += (window[aux_row][aux_col])*MaskRow[(i*3)+j];
            aux_col += 1;
        }

        aux_row += 1;
        aux_col = tcol - 1 ;

    }

	
    aux_row = trow - 1, aux_col = tcol - 1;

    for (int i = 0; i < 3; ++i){
        for (int j = 0; j < 3; ++j){
                tmpC += window[aux_row][aux_col]*MaskCol[(i*3)+j];
                aux_col += 1;
        }
        aux_row += 1;
        aux_col = tcol - 1 ;
    }

	if(threadIdx.y == 31 && blockIdx.y == 2){
		printf(" %d ",tmpC);	
	}

    d_imagefiltered[(row * width) + col] = clamp(sqrt(pow(tmpC,2) + pow(tmpR , 2)));
}

__global__ void imgGray(unsigned char * d_image, unsigned char* d_imagegray, int width, int height){

    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if ((width > col) && (height > row)){
        d_imagegray[row*width+col]=d_image[(row*width+col)*3+2]*0.3+d_image[(row*width+col)*3+1]*0.6+d_image[(row*width+col)*3]*0.2;
    }
}

int main(int argc, char const *argv[])
{
    uchar *h_image, *d_image, *h_imagegray, *d_imagegray, *h_imagefiltered, *d_imagefiltered;
    clock_t start, end; // Medir tiempos

    Mat image = imread(argv[1],1);
    Size s = image.size();
    int sizeRGB = s.width*s.height*image.channels()*sizeof(unsigned char);
    int sizeGray = s.width*s.height*sizeof(unsigned char);
    int blockSize = 32;

    h_imagegray = (unsigned char*)malloc(sizeGray);
    hipMalloc((void**)&d_image,sizeRGB);
    hipMalloc((void**)&d_imagegray,sizeGray);
    hipMalloc((void**)&d_imagefiltered,sizeGray);

    h_image = image.data;
    char h_sobelMaskRow[] = { 1 ,0, -1, 2, 0, -2, 1, 0, -1 };
    char h_sobelMaskCol[] = { -1 , -2, -1, 0, 1, 0, 1, 1, 1};

    hipMemcpyToSymbol(HIP_SYMBOL(MaskRow),h_sobelMaskRow,sizeof(char)*9);
    hipMemcpyToSymbol(HIP_SYMBOL(MaskCol),h_sobelMaskCol,sizeof(char)*9);

    hipMemcpy(d_image,h_image,sizeRGB,hipMemcpyHostToDevice);
    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimTrheads(ceil(s.width/float(blockSize)),ceil(s.height/float(blockSize)),1);

    start = clock(); //Inicia reloj
    imgGray<<<dimTrheads,dimBlock>>>(d_image,d_imagegray,s.width,s.height);
    hipDeviceSynchronize();
    hipMemcpy(h_imagegray,d_imagegray,sizeGray,hipMemcpyDeviceToHost);

    sobelFilter<<<dimTrheads,dimBlock>>>(d_imagegray,d_imagefiltered,s.width,s.height);
    hipDeviceSynchronize();
    h_imagefiltered = (unsigned char*)malloc(sizeGray);
    hipMemcpy(h_imagefiltered,d_imagefiltered,sizeGray,hipMemcpyDeviceToHost);
    end = clock(); //Finaliza reloj

    Mat imageGray;
    imageGray.create(s.height,s.width,CV_8UC1);
    imageGray.data = h_imagegray;
    imwrite("./ImageG.jpg",imageGray);

    Mat imageSobel;
    imageSobel.create(s.height,s.width,CV_8UC1);
    imageSobel.data = h_imagefiltered;
    imwrite("./ImageS.jpg",imageSobel);

    double time = ((double)(end-start))/CLOCKS_PER_SEC;
    printf("Tiempo: %lf \n", time);


 //liberar memoria 

    //free(h_image);
    //free(h_imagegray);
    //free(h_imagefiltered);
	

    hipFree(d_image);
    hipFree(d_imagegray);
    hipFree(d_imagefiltered);

    return 0;
}