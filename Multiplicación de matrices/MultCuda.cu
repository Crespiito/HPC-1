#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>


__global__ void MultiplicarMatrices(float *m1, float *m2, float *mr, int columna1, int fila1, int columna2, int fila2)
{
    int fila_r = blockIdx.y*blockDim.y+threadIdx.y;
    int columna_r = blockIdx.x*blockDim.x+threadIdx.x;
    float tmp_mult = 0;

    if ((fila < fila2) && (columna < columna1)) {
        for (int i = 0; i < fila2 ; i++) {
            tmp_mult += m1[i+columna1*fila_r]*m2[i*columna2+columna_r]; 
        }

        mr[fila_r*columna2+columna_r]= tmp_mult
    }
}

float* LlenaMatriz(int fila,int columna, FILE *archivo, float *matriz){

	for (int i = 0; i < (fila*columna); i++) {
		fscanf(archivo,"%f,",&matriz[i]);
	}
	return matriz;
}

int main(int argc, char const *argv[]) {
	
    FILE *archivo1;
	FILE *archivo2;
	int fila1, columna1, fila2, columna2, blockSize = 32, gridSize , numOper;
	// Matrices entrada Host
    float *h_m1, *h_m2;
    // Matriz salida Host
    float *h_mr;
 
    // Matrices entrada Device
    float *d_m1, *d_m2;
    // Matriz de salida Device
    float *d_mr;

	archivo1 = fopen(argv[1],"r");
	archivo2 = fopen(argv[2],"r");

	if (archivo1 != NULL && archivo2 != NULL) {
		fscanf(archivo1,"%d",&fila1);
		fscanf(archivo1,"%d",&columna1);
		fscanf(archivo2,"%d",&fila2);
		fscanf(archivo2,"%d",&columna2);

		if (columna1 == fila2) {

			// Número de operaciones por hacer
			numOper = columna1*fila2;

			gridSize= (int) ceil(numOper/blockSize);

			// Reservando y llenado de la matriz 2
			h_m1 = malloc((fila1*columna1)*sizeof(float*)); // Reserva memoria en el host
			hipMalloc(&d_m1, (fila1*columna1)); // Reserva memoria en el device
			h_m1 = LlenaMatriz(fila1,columna1,archivo1,h_m1); // Llena vector-matriz en el host
			hipMemcpy( d_m1, h_m1, (fila1*columna1), hipMemcpyHostToDevice); // Llenar vector-matriz en el device

			// Reservando y llenado de la matriz 2
			h_m2 = malloc((fila2*columna2)*sizeof(float*)); // Reserva memoria en el host
			hipMalloc(&d_m2, (fila2*columna2)); // Reserva memoria en el device
			h_m2 = LlenaMatriz(fila2,columna2,archivo2,h_m2); // Llena vector-matriz en el host
			hipMemcpy( d_m2, h_m2, (fila2*columna2), hipMemcpyHostToDevice); // Llenar vector-matriz en el device
  
			// Multiplicación de matrices
            dim3 dimBlock(blockSize,blockSize,1)
            dim3 dimThreads(gridSize,gridSize,1)
			MultiplicarMatrices<<<dimBlock, dimThreads>>>(d_m1,d_m2,d_mr, columna1, fila1, columna2 fila2);
            hipMemcpy(h_m1,h_m2,(columna1*fila2),hipMemcpyDeviceToHost);
		}
	}
}
