#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>


__global__ void MultiplicarMatrices(float *m1, float *m2, float *mr, int n, int columna2, int fila2)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	float resultado;

	for (int i = 0; i <columna2; ++i)
	{
		for (int j = 0; j < fila2; ++j)
		{
			/* code */
		}

	}

}

float* LlenaMatriz(int fila,int columna, FILE *archivo, float *matriz){

	for (int i = 0; i < fila; i++) {
		fscanf(archivo,"%f,",&matriz[i]);
	}
	return matriz;
}

int main(int argc, char const *argv[])
{
	FILE *archivo1;
	FILE *archivo2;
	int fila1, columna1, fila2, columna2, blockSize = 1024, gridSize , numOper;

	// Matrices entrada Host
    float *h_m1, *h_m2;
    // Matriz salida Host
    float *h_mr;
 
    // Matrices entrada Device
    float *d_m1, *d_m2;
    // Matriz de salida Device
    float *d_mr;

	archivo1 = fopen(argv[1],"r");
	archivo2 = fopen(argv[2],"r");

	if (archivo1 != NULL && archivo2 != NULL) {
		fscanf(archivo1,"%d",&fila1);
		fscanf(archivo1,"%d",&columna1);
		fscanf(archivo2,"%d",&fila2);
		fscanf(archivo2,"%d",&columna2);

		if (columna1 == fila2) {

			// Número de operaciones por hacer
			numOper = columna1*fila2;

			gridSize= (int) ceil(numOper/blockSize);

			// Reservando y llenado de la matriz 2
			h_m1 = malloc((fila1*columna1)*sizeof(float*)); // Reserva memoria en el host
			hipMalloc(&d_m1, (fila1*columna1)); // Reserva memoria en el device
			h_m1 = LlenaMatriz(fila1,columna1,archivo1,h_m1); // Llena vector-matriz en el host
			hipMemcpy( d_m1, h_m1, (fila1*columna1), hipMemcpyHostToDevice); // Llenar vector-matriz en el device

			// Reservando y llenado de la matriz 2
			h_m2 = malloc((fila2*columna2)*sizeof(float*)); // Reserva memoria en el host
			hipMalloc(&d_m2, (fila2*columna2)); // Reserva memoria en el device
			h_m2 = LlenaMatriz(fila2,columna2,archivo2,h_m2); // Llnea vector-matriz en el host
			hipMemcpy( d_m2, h_m2, (fila2*columna2), hipMemcpyHostToDevice); // Llenar vector-matriz en el device
  
			// Multiplicación de matrices
			MultiplicarMatrices<<<gridSize, blockSize>>>(d_m1,d_m2,d_mr,numOper, columna2, fila2);
		}
	}
}