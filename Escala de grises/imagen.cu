#include "hip/hip_runtime.h"
#include <cv.h>
#include <hip/hip_runtime.h>

using namespace cv;

__global__ void imgGray(unsigned char * d_image, unsigned char* d_imagegray, int width, int height){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if (width > col && height > row){
        d_imagegray[row*width+col]=d_imagegray[(row*width+col)*3+2]*0.21+d_imagegray[(row*width+col)*3+1]*0.71+d_imagegray[(row*width+col)*3]*0.07;
    }
}

int main(int argc, char const *argv[])
{
    char *h_image, *d_image, *h_imagegray, *d_imagegray;

    Mat image = imread(argv[1],0);
    Size s = image.size();
    int sizeRGB = s.width*s.height*image.channels(); 
    int sizeGray = s.width*s.height;
    int blocksize = 32;

    if (image.empty()){
        printf("Not found the image \n");
    }

    h_image = (unsigned char*)malloc(sizeRGB);
    h_imagegray = (unsigned char*)malloc(sizeGray);
    hipMalloc((void**)&d_image,sizeRGB);
    hipMalloc((void**)&d_imagegray,sizeGray);

    h_image = image.data;

    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimTrheads(ceil(s.width/float(blockSize)),ceil(s.height/float(blockSize)),1);
    imgGray<<<dimBlock,dimTrheads>>>(d_image,d_imagegray,s.width,s.height);
    hipDeviceSynchronize();
    hipMemcpy(h_imagegray,d_imagegray,sizeGray,hipMemcpyDeviceToHost);

    Mat imageGray;
    imageGray.create(height,width,CV_8UC1);
    imageGray.data = h_imagegray;

    namedWindow("Gray image", WINDOW_AUTOSIZE);
    imshow(argv[1],image);
    return 0;
}

