#include "hip/hip_runtime.h"
#include <cv.h>
#include <hip/hip_runtime.h>

using namespace cv;

__global__ void imgchange(){

	int col = blockIdx.x*blockDim.x+threadIdx.x;
	int row = blockIdx.y*blockDim.y+threadIdx.y;
}

int main(int argc, char const *argv[])
{
	char *h_image, *d_image, *h_imagechange, *d_imagechange;

	Mat image = imread(argv[1],0);

	if (image.empty()){
		printf("No se puede cargar la imagen \n");
	}

	//h_image = (unsigned char*)malloc();

	Size s = image.size();
	namedWindow(argv[1], WINDOW_AUTOSIZE);
	imshow(argv[1],image);
	return 0;
}

