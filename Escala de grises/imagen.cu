#include "hip/hip_runtime.h"
#include <cv.h>
#include <hip/hip_runtime.h>

using namespace cv;

__global__ void imgGray(){

}

int main(int argc, char const *argv[])
{
	char *h_image, *d_image, *h_imagegray, *d_imagegray;

	Mat image = imread(argv[1],0);
	Size s = image.size();
	int sizeRGB = s.width*s.height*image.channels(); 
	int sizeGray = s.width*s.height;
	int blocksize = 32, gridSize = ;

	if (image.empty()){
		printf("Not found the image \n");
	}

	h_image = (unsigned char*)malloc(sizeRGB);
	h_imagegray = (unsigned char*)malloc(sizeGray);
	hipMalloc((void**)&d_image,sizeRGB);
	hipMalloc((void**)&d_imagegray,sizeGray);

	h_image = image.data;

	namedWindow("Gray image", WINDOW_AUTOSIZE);
	imshow(argv[1],image);
	return 0;
}

