#include "hip/hip_runtime.h"
#include "opencv2/opencv.hpp"
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace cv;

__global__ void imgGray(unsigned char * d_image, unsigned char* d_imagegray, int width, int height){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if ((width > col) && (height > row)){
        d_imagegray[row*width+col]=d_image[(row*width+col)*3+2]*0.3+d_image[(row*width+col)*3+1]*0.6+d_image[(row*width+col)*3]*0.2;
    }
}

int main(int argc, char const *argv[])
{
    uchar *h_image, *d_image, *h_imagegray, *d_imagegray;

    Mat image = imread(argv[1],1);
    Size s = image.size();
    int sizeRGB = s.width*s.height*image.channels()*sizeof(unsigned char);
    int sizeGray = s.width*s.height*sizeof(unsigned char);
    int blockSize = 32;
    printf("%d , %d \n",sizeRGB, sizeGray);
    if (image.empty()){
        printf("Not found the image \n");
    }

    h_imagegray = (unsigned char*)malloc(sizeGray);
    hipMalloc((void**)&d_image,sizeRGB);
    hipMalloc((void**)&d_imagegray,sizeGray);

    h_image = image.data;

    hipMemcpy(d_image,h_image,sizeRGB,hipMemcpyHostToDevice);
    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimTrheads(ceil(s.width/float(blockSize)),ceil(s.height/float(blockSize)),1);
    imgGray<<<dimTrheads,dimBlock>>>(d_image,d_imagegray,s.width,s.height);
    hipDeviceSynchronize();
    hipMemcpy(h_imagegray,d_imagegray,sizeGray,hipMemcpyDeviceToHost);

    Mat imageGray;
    imageGray.create(s.height,s.width,CV_8UC1);
    imageGray.data = h_imagegray;
    imwrite("./ImageG.jpg",imageGray);

    return 0;
}

